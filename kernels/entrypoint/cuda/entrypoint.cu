#include "hip/hip_runtime.h"
#include "kernels/entrypoint/cuda/entrypoint.hpp"
#include "kernels/kernels.hpp"

namespace nova {

void kernel_raytrace(dim3 num_blocks,
                     dim3 block_size,
                     const SceneParams& scene_params,
                     uint time,
                     hipSurfaceObject_t temp_color1,
                     uint2 pixel_dims,
                     TriangleData* triangles,
                     TriangleMetaData* tri_meta,
                     FlatBVHNode* bvh,
                     AreaLightData* lights,
                     uint num_lights,
                     hipTextureObject_t materials,
                     hipTextureObject_t sky,
                     uint denoise_available,
                     hipSurfaceObject_t albedo_feature1,
                     hipSurfaceObject_t normal_feature1) {
  kernel_raytrace<<<num_blocks, block_size>>>(
    scene_params, time, temp_color1, pixel_dims, triangles, tri_meta, bvh, lights, num_lights,
    materials, sky, denoise_available, albedo_feature1, normal_feature1);
}

void kernel_accumulate(dim3 num_blocks,
                       dim3 block_size,
                       int sample_index,
                       uint denoise_available,
                       hipTextureObject_t temp_color1,
                       hipTextureObject_t albedo_feature1,
                       hipTextureObject_t normal_feature1,
                       hipTextureObject_t prev_color,
                       hipTextureObject_t prev_albedo_feature,
                       hipTextureObject_t prev_normal_feature,
                       hipSurfaceObject_t temp_color2,
                       hipSurfaceObject_t albedo_feature2,
                       hipSurfaceObject_t normal_feature2,
                       uint2 pixel_dims) {
  kernel_accumulate<<<num_blocks, block_size>>>(
    sample_index, denoise_available, temp_color1, albedo_feature1, normal_feature1, prev_color,
    prev_albedo_feature, prev_normal_feature, temp_color2, albedo_feature2, normal_feature2,
    pixel_dims);
}

void kernel_post_process(dim3 num_blocks,
                         dim3 block_size,
                         const SceneParams& scene_params,
                         hipTextureObject_t temp_color2,
                         hipSurfaceObject_t pixels,
                         uint2 pixel_dims) {
  kernel_post_process<<<num_blocks, block_size>>>(scene_params, temp_color2, pixels, pixel_dims);
}

}