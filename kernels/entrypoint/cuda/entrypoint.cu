#include "hip/hip_runtime.h"
#include "kernels/entrypoint/cuda/entrypoint.hpp"
#include "kernels/kernels.hpp"

namespace nova {

void kernel_raytrace(dim3 num_blocks,
                     dim3 block_size,
                     const SceneParams& scene_params,
                     int sample_num,
                     hipSurfaceObject_t temp_pixels1,
                     hipSurfaceObject_t temp_pixels2,
                     uint2 pixel_dims,
                     TriangleData* triangles,
                     TriangleMetaData* tri_meta,
                     FlatBVHNode* bvh,
                     hipTextureObject_t materials,
                     hipTextureObject_t sky) {
  kernel_raytrace<<<num_blocks, block_size>>>(scene_params, sample_num, temp_pixels1, temp_pixels2,
                                              pixel_dims, triangles, tri_meta, bvh, materials, sky);
}

void kernel_interpolate(dim3 num_blocks,
                        dim3 block_size,
                        hipTextureObject_t temp_pixels1,
                        hipSurfaceObject_t temp_pixels2,
                        uint2 pixel_dims,
                        uint* rem_pixels_counter,
                        int2* rem_coords) {
  kernel_interpolate<<<num_blocks, block_size>>>(temp_pixels1, temp_pixels2, pixel_dims,
                                                 rem_pixels_counter, rem_coords);
}

void kernel_fill_remaining(dim3 num_blocks,
                           dim3 block_size,
                           const SceneParams& scene_params,
                           int sample_num,
                           hipSurfaceObject_t temp_pixels2,
                           uint2 pixel_dims,
                           TriangleData* triangles,
                           TriangleMetaData* tri_meta,
                           FlatBVHNode* bvh,
                           hipTextureObject_t materials,
                           hipTextureObject_t sky,
                           uint* rem_pixels_counter,
                           int2* rem_coords) {
  kernel_fill_remaining<<<num_blocks, block_size>>>(scene_params, sample_num, temp_pixels2,
                                                    pixel_dims, triangles, tri_meta, bvh, materials,
                                                    sky, rem_pixels_counter, rem_coords);
}

void kernel_post_process(dim3 num_blocks,
                         dim3 block_size,
                         const SceneParams& scene_params,
                         int sample_num,
                         hipTextureObject_t temp_pixels2,
                         hipTextureObject_t prev_pixels,
                         hipSurfaceObject_t pixels,
                         uint2 pixel_dims) {
  kernel_post_process<<<num_blocks, block_size>>>(scene_params, sample_num, temp_pixels2,
                                                  prev_pixels, pixels, pixel_dims);
}

}