#include "hip/hip_runtime.h"
#include "texture.h"
#include "transforms.h"
#include "vector_math.h"
#include "matrix.h"
#include "hip/hip_math_constants.h"
#include "constants.h"

__device__
float3 read_material(hipTextureObject_t materials, const TriangleMetaData& meta,
                     float2 texture_coord, int index, float3 default_material) {
  if (meta.diffuse_index == -1 && meta.metallic_index == -1 && meta.roughness_index == -1 &&
      meta.ambient_occlusion_index == -1 && meta.normal_index == -1) {
    return default_material;
  }
  if (index == -1) {
    return default_material;
  }

  uchar3 texture = make_uchar3(
    tex2DLayered<uchar4>(materials, texture_coord.x, texture_coord.y, index));
  return uint3_to_float3(make_float3(texture));
}

__device__
float3 compute_normal(hipTextureObject_t materials, const TriangleMetaData& meta,
                      float2 texture_coord, float3 barycentric) {
  // Interpolate triangle normal from vertex data
  float3 normal = normalize(
    triangle_interpolate(barycentric, meta.normal1, meta.normal2, meta.normal3)
  );

  // Use the normal map to compute pixel normal if it exists
  if (meta.normal_index != -1) {
    float3 tangent = normalize(
      triangle_interpolate(barycentric, meta.tangent1, meta.tangent2, meta.tangent3)
    );
    float3 bitangent = normalize(
      triangle_interpolate(barycentric, meta.bitangent1, meta.bitangent2, meta.bitangent3)
    );

    // Create TBN matrix and use it to convert tangent space pixel normal to world space
    Mat3x3 tbn = transpose({
      tangent,
      bitangent,
      normal
    });

    float3 pixel_normal = 
      read_material(materials, meta, texture_coord, meta.normal_index, make_float3(0.0f));
    pixel_normal = normalize(pixel_normal * 2.0f - 1.0f);
    normal = normalize(tbn * pixel_normal);
  }

  return normal;
}

__device__
float3 fresnel_schlick(float cos_theta, float3 f0) {
  float a = 1.0f - cos_theta;
  float a2 = a * a;
  float a5 = a2 * a2 * a;
  return f0 + (1.0f - f0) * a5;
}

__device__
float distribution_ggx(float n_dot_h, float roughness) {
  float a = roughness * roughness;
  float a2 = a * a;

  float denom = n_dot_h * n_dot_h * (a2 - 1.0f) + 1.0f;
  denom = HIP_PI * denom * denom;

  return a2 / denom;
}

__device__
float geometry_smith(float n_dot_v, float n_dot_l, float nvl, float roughness) {
  float r = roughness + 1.0f;
  float k = r * r / 8.0f;
  float m = 1.0f - k;

  return nvl / ((n_dot_v * m + k) * (n_dot_l * m + k));
}

__device__
float3 specularity(float3 view_dir, float3 half_dir, float3 diffuse, float metallic) {
  float h_dot_v = max(dot(half_dir, view_dir), 0.0f);
  float3 f0 = mix(make_float3(0.04f), diffuse, metallic);
  // fresnel equation
  float3 f = fresnel_schlick(h_dot_v, f0);

  return f;
}

__device__
float3 shade(float3 light_dir, float3 view_dir, float3 half_dir, float light_distance, 
             float3 normal, float3 diffuse, float3 kS, float metallic, float roughness) {
  float n_dot_v = max(dot(normal, view_dir), 0.0f);
  float n_dot_l = max(dot(normal, light_dir), 0.0f);
  float n_dot_h = max(dot(normal, half_dir), 0.0f);
  
  float nvl = n_dot_v * n_dot_l;

  // normal distribution function
  float d = distribution_ggx(n_dot_h, roughness);
  // geometry function
  float g = geometry_smith(n_dot_v, n_dot_l, nvl, roughness);

  // diffuse
  float3 kD = (1.0f - kS) * (1.0f - metallic);

  float3 brdf = kD * diffuse * CUDART_INV_PI + d * kS * g / fmaxf(4.0f * nvl, 1e-3f);
  float3 radiance = constants.light_intensity / fmaxf(light_distance * light_distance, 1.0f);

  return brdf * radiance * n_dot_l;
}
