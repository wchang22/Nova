#include "raytrace.h"

void kernel_raytrace(const Dims& global_dims,
                     const KernelConstants& kernel_constants,
                     hipTextureObject_t image_out,
                     EyeCoords ec,
                     TriangleData* triangles,
                     TriangleMetaData* tri_meta,
                     FlatBVHNode* bvh,
                     hipTextureObject_t materials) {
  
}