#include "hip/hip_runtime.h"
#include "raytrace.h"
#include "constants.h"
#include "vector_math.h"
#include "intersection.h"
#include "texture.h"
#include "transforms.h"
#include "types.h"
#include "backend/cuda/types/types.h"

__device__
bool find_intersection(
  TriangleData* triangles,
  FlatBVHNode* bvh,
  const Ray& ray,
  Intersection& min_intrs,
  bool fast
) {
  /*
  * We maintain a double ended stack for space efficiency.
  * BVHNodes are pushed from the front to the back of the stack and
  * triangle offsets and nums are pushed from the back to the front of the stack.
  * This allows work items to find more than one leaf node before searching for
  * triangles and reduces branch divergence.
  */
  uint stack[STACK_SIZE];
  int node_ptr = -1;
  int tri_ptr = STACK_SIZE;

  // Set first value of stack to 0. We stop traversing when we pop this value.
  stack[++node_ptr] = 0;

  int node_index = 0;
  do {
    do {
      FlatBVHNode node = bvh[node_index];

      if (!intersects_aabb(ray, make_float3(node.top_offset_left),
                                make_float3(node.bottom_num_right))) {
        node_index = stack[node_ptr--];
        continue;
      }

      // Inner node, no triangles
      if (node.bottom_num_right.w >= 0) {
        // Traverse left and right children
        uint left = node.top_offset_left.w;
        uint right = node.bottom_num_right.w;

        if (!left && !right) {
          node_index = stack[node_ptr--];
        } else {
          node_index = left ? left : right;
          if (left && right) {
            stack[++node_ptr] = right;
          }
        }
      }
      // Leaf node, no children
      else {
        uint offset = node.top_offset_left.w;
        uint num = -node.bottom_num_right.w;

        // Pack offset and num into a single uint to save memory
        uint packed_triangle_data =
          (offset & constants.triangle_offset_mask) | (num << constants.triangle_num_shift);

        // Push list of triangles to stack back
        stack[--tri_ptr] = packed_triangle_data;

        node_index = stack[node_ptr--];
      }
    // Make sure tri_ptr and node_ptr do not collide
    } while (node_index && tri_ptr > node_ptr + 2);

    while (tri_ptr < STACK_SIZE) {
      // Pop list of triangles from stack back
      uint packed_triangle_data = stack[tri_ptr++];

      uint offset = packed_triangle_data & constants.triangle_offset_mask;
      uint num = packed_triangle_data >> constants.triangle_num_shift;

      // If intersected, compute intersection for all triangles in the node
      for (uint i = offset; i < offset + num; i++) {
        if (intersects_triangle(ray, min_intrs, i, triangles[i]) && fast) {
          return true;
        }
      }
    }
  } while (node_index);

  return min_intrs.tri_index != -1;
}

__device__
float3 trace_ray(
  uint2 pixel_coords,
  EyeCoords ec,
  TriangleData* triangles,
  TriangleMetaData* tri_meta,
  FlatBVHNode* bvh,
  hipTextureObject_t materials
) { 
  float2 alpha_beta = ec.coord_scale * (make_float2(pixel_coords) - ec.coord_dims + 0.5f);
  float3 ray_dir = normalize(alpha_beta.x * ec.eye_coord_frame.x -
                             alpha_beta.y * ec.eye_coord_frame.y -
                                            ec.eye_coord_frame.z);
  float3 ray_pos = ec.eye_pos;

  float3 color = make_float3(0.0f);
  float3 reflectance = make_float3(1.0f);

  for (int depth = 0; depth < constants.ray_recursion_depth; depth++) {
    Ray ray = create_ray(ray_pos, ray_dir, RAY_EPSILON);

    Intersection intrs = no_intersection();

    // Cast primary/reflection ray
    if (!find_intersection(triangles, bvh, ray, intrs, false)) {
      break;
    }

    TriangleMetaData meta = tri_meta[intrs.tri_index];

    // Calculate intersection point
    float3 intrs_point = ray.origin + ray.direction * intrs.length;

    // Interpolate texture coords from vertex data
    float2 texture_coord = triangle_interpolate(
      intrs.barycentric, meta.texture_coord1, meta.texture_coord2, meta.texture_coord3
    );

    // Look up materials
    float3 diffuse = read_material(materials, meta, texture_coord, meta.diffuse_index,
                                   constants.default_diffuse);
    float metallic = read_material(materials, meta, texture_coord, meta.metallic_index,
                                   make_float3(constants.default_metallic)).x;
    float roughness = read_material(materials, meta, texture_coord, meta.roughness_index,       
                                    make_float3(constants.default_roughness)).x;
    float ambient_occlusion = read_material(materials, meta, texture_coord,   
                                            meta.ambient_occlusion_index,
                                            make_float3(constants.default_ambient_occlusion)).x;

    float3 normal = compute_normal(materials, meta, texture_coord, intrs.barycentric);

    // Add ambient color even if pixel is in shadow
    float3 intrs_color = diffuse * ambient_occlusion * 0.03f;

    // Calculate lighting params
    float3 light_dir = normalize(constants.light_position - intrs_point);
    float3 view_dir = normalize(ec.eye_pos - intrs_point);
    float3 half_dir = normalize(light_dir + view_dir);
    float light_distance = distance(constants.light_position, intrs_point);
    float3 kS = specularity(view_dir, half_dir, diffuse, metallic);

    // Cast a shadow ray to the light
    Ray shadow_ray = create_ray(intrs_point, light_dir, RAY_EPSILON);
    Intersection light_intrs = no_intersection();
    // Ensure objects blocking light are not behind the light
    light_intrs.length = light_distance;

    // Shade the pixel if ray is not blocked
    if (!find_intersection(triangles, bvh, shadow_ray, light_intrs, true)) {
      intrs_color += shade(light_dir, view_dir, half_dir, light_distance,
                          normal, diffuse, kS, metallic, roughness);
    }

    /*
    * Normally, color is calculated recursively:
    * (intrs_color + specular * (intrs_color of reflected ray))
    * So we use an additional "reflectance" value to unroll the recursion
    */
    color += reflectance * intrs_color;
    reflectance *= kS;

    // Stop if reflectance is too low to produce a color change
    if (all(isless(reflectance, COLOR_EPSILON))) {
      break;
    }

    // Reflect ray off of intersection point
    ray_pos = intrs_point;
    ray_dir = reflect(ray_dir, normal);
  }

  return clamp(color, 0.0f, 1.0f);
}

__global__
void raytrace(
  uchar4* pixels,
  uint2 pixel_dims,
  EyeCoords ec,
  TriangleData* triangles,
  TriangleMetaData* tri_meta,
  FlatBVHNode* bvh,
  hipTextureObject_t materials
) {
  uint2 pixel_coords = { blockDim.x * blockIdx.x + threadIdx.x,
                         blockDim.y * blockIdx.y + threadIdx.y };
  pixel_coords.y = 2 * pixel_coords.y + (pixel_coords.x & 1);

  float3 color = trace_ray(pixel_coords, ec, triangles, tri_meta, bvh, materials);

  int pixel_index = linear_index(make_int2(pixel_coords), pixel_dims.x);
  pixels[pixel_index] = make_uchar4(make_float4(color, 1.0f) * 255.0f);
}

__global__
void interpolate(
  uchar4* pixels,
  uint2 pixel_dims,
  EyeCoords ec,
  TriangleData* triangles,
  TriangleMetaData* tri_meta,
  FlatBVHNode* bvh,
  hipTextureObject_t materials
) {
  uint2 pixel_coords = { blockDim.x * blockIdx.x + threadIdx.x,
                         blockDim.y * blockIdx.y + threadIdx.y };
  pixel_coords.y = 2 * pixel_coords.y + 1 - (pixel_coords.x & 1);

  // Sample 4 neighbours
  int2 neighbor_offsets[] = { { 0, -1 }, { -1, 0 }, { 1, 0 }, { 0, 1 } };
  uint4 neighbors[4];
  for (uint i = 0; i < 4; i++) {
    int2 offset = neighbor_offsets[i];
    int index = linear_index(make_int2(
      clamp((int) pixel_coords.x + offset.x, 0, (int) pixel_dims.x - 1),
      clamp((int) pixel_coords.y + offset.y, 0, (int) pixel_dims.y - 1)
    ), pixel_dims.x);
    neighbors[i] = make_uint4(pixels[index]);
  }

  // Check color differences in the neighbours
  uint4 color_max = max(neighbors[0], max(neighbors[1], max(neighbors[2], neighbors[3])));
  uint4 color_min = min(neighbors[0], min(neighbors[1], min(neighbors[2], neighbors[3])));
  float3 color_range = make_float3(make_uint3(color_max - color_min)) / 255.0f;

  uchar4 color;
  // If difference is large, raytrace to find color
  if (length(color_range) > INTERP_THRESHOLD) {
    float3 rt_color = trace_ray(pixel_coords, ec, triangles, tri_meta, bvh, materials);
    color = make_uchar4(make_float4(rt_color, 1.0f) * 255.0f);
  }
  // Otherwise, interpolate
  else {
    color = make_uchar4((neighbors[0] + neighbors[1] + neighbors[2] + neighbors[3]) / 4U);
  }
  
  int pixel_index = linear_index(make_int2(pixel_coords), pixel_dims.x);
  pixels[pixel_index] = color;
}

void kernel_raytrace(
  uint3 global_dims,
  const KernelConstants& kernel_constants,
  uchar4* pixels,
  uint2 pixel_dims,
  EyeCoords ec,
  TriangleData* triangles,
  TriangleMetaData* tri_meta,
  FlatBVHNode* bvh,
  hipTextureObject_t materials
) {
  dim3 block_size { 8, 8, 1 };
  dim3 num_blocks { global_dims.x / block_size.x, global_dims.y / block_size.y, 1 };
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(constants), &kernel_constants,
                                sizeof(KernelConstants), 0, hipMemcpyHostToDevice));
  raytrace<<<num_blocks, block_size>>>(pixels, pixel_dims, ec, triangles, tri_meta, bvh, materials);
}

void kernel_interpolate(
  uint3 global_dims,
  const KernelConstants& kernel_constants,
  uchar4* pixels,
  uint2 pixel_dims,
  EyeCoords ec,
  TriangleData* triangles,
  TriangleMetaData* tri_meta,
  FlatBVHNode* bvh,
  hipTextureObject_t materials
) {
  dim3 block_size { 8, 8, 1 };
  dim3 num_blocks { global_dims.x / block_size.x, global_dims.y / block_size.y, 1 };
  CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(constants), &kernel_constants,
                                sizeof(KernelConstants), 0, hipMemcpyHostToDevice));
  interpolate<<<num_blocks, block_size>>>(pixels, pixel_dims, ec, triangles,
                                          tri_meta, bvh, materials);
}