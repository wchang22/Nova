#include "hip/hip_runtime.h"
#include "raytrace.h"
#include "vector_conversions.h"

__global__
void raytrace(const KernelConstants& kernel_constants,
              hipSurfaceObject_t image_out,
              EyeCoords ec,
              TriangleData* triangles,
              TriangleMetaData* tri_meta,
              FlatBVHNode* bvh,
              hipTextureObject_t materials) {
  uint2 pixel_coords = { blockDim.x * blockIdx.x + threadIdx.x,
                         blockDim.y * blockIdx.y + threadIdx.y };
  float3 color = { 0.0f, 0.0f, 0.0f };
  
  

  uchar4 image_color = convert_uchar4(make_float4(color, 1.0f) * 255.0f);
  surf2Dwrite(image_color, image_out, pixel_coords.x * sizeof(uchar4), pixel_coords.y);
}

void kernel_raytrace(const Dims& global_dims,
                     const KernelConstants& kernel_constants,
                     hipSurfaceObject_t image_out,
                     EyeCoords ec,
                     TriangleData* triangles,
                     TriangleMetaData* tri_meta,
                     FlatBVHNode* bvh,
                     hipTextureObject_t materials) {
  dim3 num_blocks { std::get<0>(global_dims), std::get<1>(global_dims), std::get<2>(global_dims) };
  dim3 block_size { 1, 1, 1 };
  raytrace<<<num_blocks, block_size>>>(kernel_constants, image_out, ec, triangles,
                                       tri_meta, bvh, materials);
}